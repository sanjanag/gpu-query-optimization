#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>

using namespace std;

#define N 2048*2048
#define THREADS_PER_BLOCK 512

__global__ void add(int* a, int* b, int* c){

	int index= threadIdx.x + blockDim.x*blockIdx.x;
	c[index] = a[index] + b[index];

}


int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int)*N;
	
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	for(int i=0;i<N;i++)
		a[i]=rand();

	for(int i=0;i<N;i++)
		b[i]=rand();

	hipMalloc((void**)&d_a, size); 
	hipMalloc((void**)&d_b, size); 	
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a,d_b,d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a);
	free(b);
	free(c);
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c); 
	
	return 0;	

}
